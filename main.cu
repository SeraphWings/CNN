#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE
#include "mnist.h"
#include "layer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <time.h>
#include <cassert>
#include <math.h>
#include <stdio.h>   
#include <stdlib.h> 

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;

int train_onehot[60000][10];
int test_onehot[10000][10];
double input_kernel[26*26][343];
double output_kernel[343][10];
double input_bias[343];
double output_bias[10];
double error;
double L_rate = 1.0e-5f;

// Define layers of CNN
static Layer train_input = Layer(1, 28, 28);
static Layer conved = Layer(1, 27, 27);
static Layer maxpooled = Layer(1, 26, 26);
static Layer flattenned = Layer(1, 1, 26*26);
static Layer hidden = Layer(1, 1, 343);
static Layer densed = Layer(1, 1, 10);

static void learn();
static unsigned int classify(double data[28][28], int cnt);
double test_on_train();
double test_on_test();
static double forward_pass(double data[28][28], int cnt);
static double back_pass(int cnt);


static inline void loaddata()
{
	mnist_load("./data/train-images.idx3-ubyte", "./data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("./data/t10k-images.idx3-ubyte", "./data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);

}

/*
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
*/

static void printimg(double data[28][28]){
	for (int i = 0; i < 28; ++i)
	{
		for (int j = 0; j < 28; ++j)
		{
			printf("%.2lf ", data[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void train_one_hot(mnist_data *data, int output[60000][10]){
	for(int i = 0;i < 60000;i++){
		for(int j = 0;j < 10;j++){
			if(data[i].label == j){
				output[i][j] = 1;
			}
			else{
				output[i][j] = 0;
			}
			
		}
	}
}

void test_one_hot(mnist_data *data, int output[10000][10]){
	for(int i = 0;i < 10000;i++){
		for(int j = 0;j < 10;j++){
			if(data[i].label == j){
				output[i][j] = 1;
			}
			else{
				output[i][j] = 0;
			}
			
		}
	}
}


void generateKernel(){

	for (int i = 0; i < 26*26; i++)
	{
		for (int j = 0; j < 343; j++)
		{
			input_kernel[i][j] = ( ( rand() + -1 * rand() )%10) * 0.1;

		}

	}

	for (int i = 0; i < 343; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			output_kernel[i][j] = ( ( rand() + -1 * rand() )%10) * 0.1;

		}

	}

}

void generateBias(){

		for (int i = 0; i < 343; i++)
		{
			input_bias[i] = ( ( rand() + -1 * rand() )%10) *0.1;

		}

		for (int i = 0; i < 10; i++)
		{
			output_bias[i] = ( ( rand() + -1 * rand() )%10) *0.1;

		}

}



double crossEntropy(int cnt){
	//printf("cross entropy\n");
	double output = 0.0;
	for (int i = 0; i < 10; i++)
	{
		//printf("%.2lf ", densed.data1D[i]);
		if(densed.data1D[i] > 0){
			
			output += log( densed.data1D[i] ) * train_onehot[cnt][i];
		}
		else if(densed.data1D[i] == 0.0){
			output += log( 1.0e-4f ) * train_onehot[cnt][i];
		}
		else{
			printf("%d %d log() error\n", cnt, i);
		}

	}
	//printf("\n");

	output *= -1.0;
	output /= 10.0;

	return output;
	
}

void test_flatten(){
	for (int i = 0; i < maxpooled.H; i++)
	{
		for (int j = 0; j <maxpooled.W; j++)
		{
			flattenned.data1D[i * maxpooled.W + j] = maxpooled.data2D[i][j];
		}
		
	}
}

int main(int argc, const  char **argv)
{
	loaddata();
	//printimg(train_set[0].data);
	train_one_hot(train_set, train_onehot);
	test_one_hot(test_set, test_onehot);
	srand (time(NULL));
	generateKernel();
	generateBias();

	/*
	printf("origin\n");
	train_input.readInput(train_set[0].data);
	train_input.printData();
	
	printf("convolution\n");
	conved.conv2D(train_input.data2D);
	conved.printData();
	
	printf("maxpooling\n");
	maxpooled.maxPooling(conved.data2D);
	maxpooled.printData();
	
	printf("flatten\n");
	test_flatten();
	//flattenned.flatten(maxpooled.data2D);
	flattenned.printData();

	printf("hidden\n");
	//hidden.in_hidden(flattenned.data1D, input_kernel, input_bias);
	double* device_flatten;
	if ( hipMalloc((void**)&device_flatten, sizeof(double) * 676)  != hipSuccess) printf("device_flatten error\n");
	if ( hipMemcpy(device_flatten, flattenned.data1D, sizeof(double) * 676, hipMemcpyHostToDevice) != hipSuccess) printf("device_flatten cpy error\n");
	double* device_hidden;
	if ( hipMalloc((void**)&device_hidden, sizeof(double) * 343) != hipSuccess) printf("device_hidden error\n");
	double* device_input_kernel;
	if ( hipMalloc((void**)&device_input_kernel, sizeof(double) * 676 * 343) != hipSuccess) printf("device_input_kernel error\n");
	if ( hipMemcpy(device_input_kernel, input_kernel, sizeof(double) * 676 * 343, hipMemcpyHostToDevice) != hipSuccess) printf("device_input_kernel cpy error\n");
	double* device_input_bias;
	if ( hipMalloc((void**)&device_input_bias, sizeof(double) * 343) != hipSuccess) printf("device_input_bias error\n");
	if ( hipMemcpy(device_input_bias, input_bias, sizeof(double) * 343, hipMemcpyHostToDevice) != hipSuccess) printf("device_input_bias cpy error\n");

	
	GPU_in_hidden<<<1,1>>>(device_flatten, device_hidden, device_input_kernel, device_input_bias);
	if ( hipMemcpy(hidden.data1D, device_hidden, sizeof(double) * 343, hipMemcpyDeviceToHost) != hipSuccess) printf("device_hidden cpy back error\n");
	hipDeviceSynchronize();
	hipFree(device_flatten);
	hipFree(device_input_kernel);
	hipFree(device_input_bias);
	hipFree(device_hidden);
	hidden.printData();
	
	printf("densed\n");
	densed.dense(hidden.data1D, output_kernel, output_bias);
	densed.printData();
	
	error = crossEntropy(0);
	*/

	// printf("%lf \n", forward_pass(train_set[0].data, 0));
	// back_pass(0);
	// // test_on_train();
	// printf("%lf \n", forward_pass(train_set[3].data, 3));
	// back_pass(3);
	// printimg(test_set[0].data);
	
	learn();
	
	/*
	learn();
	//test();
	*/
	return 0;
}

// Forward propagation of a single row in dataset
static double forward_pass(double data[28][28], int cnt){

	clock_t start, end;
	start = clock();

	// printf("origin\n");
	train_input.readInput(train_set[cnt].data);
	// train_input.printData();
	
	// printf("convolution\n");
	conved.conv2D(train_input.data2D);
	// conved.printData();
	
	// printf("maxpooling\n");
	maxpooled.maxPooling(conved.data2D);
	// maxpooled.printData();
	
	// printf("flatten\n");
	test_flatten();
	//flattenned.flatten(maxpooled.data2D);
	// flattenned.printData();

	// printf("hidden\n");
	// hidden.in_hidden(flattenned.data1D, input_kernel, input_bias);
	double* device_flatten;
	if ( hipMalloc((void**)&device_flatten, sizeof(double) * 676)  != hipSuccess) printf("device_flatten error\n");
	if ( hipMemcpy(device_flatten, flattenned.data1D, sizeof(double) * 676, hipMemcpyHostToDevice) != hipSuccess) printf("device_flatten cpy error\n");
	double* device_hidden;
	if ( hipMalloc((void**)&device_hidden, sizeof(double) * 343) != hipSuccess) printf("device_hidden error\n");
	double* device_input_kernel;
	if ( hipMalloc((void**)&device_input_kernel, sizeof(double) * 676 * 343) != hipSuccess) printf("device_input_kernel error\n");
	if ( hipMemcpy(device_input_kernel, input_kernel, sizeof(double) * 676 * 343, hipMemcpyHostToDevice) != hipSuccess) printf("device_input_kernel cpy error\n");
	double* device_input_bias;
	if ( hipMalloc((void**)&device_input_bias, sizeof(double) * 343) != hipSuccess) printf("device_input_bias error\n");
	if ( hipMemcpy(device_input_bias, input_bias, sizeof(double) * 343, hipMemcpyHostToDevice) != hipSuccess) printf("device_input_bias cpy error\n");

	
	GPU_in_hidden<<<1,64>>>(device_flatten, device_hidden, device_input_kernel, device_input_bias);
	if ( hipMemcpy(hidden.data1D, device_hidden, sizeof(double) * 343, hipMemcpyDeviceToHost) != hipSuccess) printf("device_hidden cpy back error\n");
	hipDeviceSynchronize();
	hipFree(device_flatten);
	hipFree(device_input_kernel);
	hipFree(device_input_bias);
	hipFree(device_hidden);
	// hidden.printData();
	
	// printf("densed\n");
	densed.dense(hidden.data1D, output_kernel, output_bias);
	// if(cnt % 1000 == 0) densed.printData();
	// densed.printData();
	
	error = crossEntropy(cnt);
	// if(cnt % 10000 == 0) printf("%d label = %d , predict = %d \n", cnt, train_set[cnt].label, classify(train_set[cnt].data, cnt));
	//if(cnt < 5) printf("cnt = %d \t error = %lf\n", cnt, error);
	//if(cnt % 1000 == 0) printf("cnt = %d \t error = %lf\n", cnt, error);
	//if(cnt % 10000 == 0) printf("cnt = %d \t error = %lf\n", cnt, error);

	end = clock();

	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

// Back propagation to update weights
static double back_pass(int cnt)
{
	//printf("back propagation\n");
	clock_t start, end;

	start = clock();

	//output delta
	double output_delta[10];
	for (int i = 0; i < 10; i++)
	{
		output_delta[i] = densed.data1D[i] - (double)train_onehot[cnt][i];
		
	}

	// printf("out delta\n");
	// for (int i = 0; i < 10; i++)
	// {
	// 	printf("%.2lf ", output_delta[i]);

	// }
	// printf("\n");

	//hidden delta
	double hidden_delta[343];
	double pre_act[343];
	for (int i = 0; i < 343; i++)
	{
		pre_act[i] = 0.0;
		for (int j = 0; j < 10; j++)
		{
			pre_act[i] += output_delta[j] * output_kernel[i][j];
		}
		
	}

	double partial[343];
	for (int i = 0; i < 343; i++)
	{
		partial[i] = 0.0;
		for (int j = 0; j < 676; j++)
		{
			partial[i] += input_kernel[j][i];
		}
			
	}
	
	for (int i = 0; i < 343; i++)
	{
		
		hidden_delta[i] = pre_act[i] * partial[i];
		
	}



	// printf("hidden delta\n");
	// for (int i = 0; i < 343; i++)
	// {
	// 	printf("%lf ", hidden_delta[i]);
		
	// }
	// printf("\n");
	

	/*
	//input delta
	double input_delta[26*26];
	for (int i = 0; i < 26*26; i++)
	{
		input_delta[i] = 0.0;
		for (int j = 0; j < 343; j++)
		{
			input_delta[i] += flattenned.data1D[i] * hidden_delta[j] * input_kernel[i][j];
		}
		
	}
	*/
	
	//output kernel&bias update
	for (int i = 0; i < 343; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			output_kernel[i][j] -= L_rate * hidden.data1D[j] * output_delta[j];
		}

	}
	// printf("output kernel\n");
	// for (int i = 0; i < 343; i++)
	// {
	// 	for (int j = 0; j < 10; j++)
	// 	{
	// 		printf("%lf ", output_kernel[i][j]);
	// 	}

	// }
	// printf("\n");

	for (int j = 0; j < 10; j++)
	{
		output_bias[j] -=   L_rate *  output_delta[j];
	}

	// printf("output bias\n");
	// for (int i = 0; i < 10; i++)
	// {
	// 	printf("%lf ", output_bias[i]);
	// }
	// printf("\n");

	// printf("input kernel before \n");
	// for (int i = 0; i < 26*26; i++)
	// {
		
	// 	printf("%lf ", input_kernel[i][0]);
	// }
	// printf("\n");

	//input kernel&bias update
	for (int i = 0; i < 26*26; i++)
	{
		for (int j = 0; j < 343; j++)
		{
			input_kernel[i][j] = input_kernel[i][j] - L_rate * flattenned.data1D[i] * hidden_delta[j];
		}

	}

	// printf("input kernel\n");
	// for (int i = 0; i < 26*26; i++)
	// {
		
	// 	printf("%lf ", input_kernel[i][0]);
	// }
	// printf("\n");

	for (int j = 0; j < 343; j++)
	{
		input_bias[j] -=   L_rate *  hidden_delta[j];
	}


	end = clock();
	return ((double) (end - start)) / CLOCKS_PER_SEC;
}

static void learn()
{
	
	int epoch = 3;
	double time_taken = 0.0;
	

	fprintf(stdout ,"Learning \n");

	while (epoch > 0) {	
		double epoch_err = 0.0;
		int train_idx;
		printf("epoch %d\n", epoch);
		for (int i = 0; i < train_cnt; i++) {
			
			//printf("forward passing\n");
			train_idx = rand()%train_cnt;
			time_taken += forward_pass(train_set[train_idx].data, train_idx);
			time_taken += back_pass(train_idx);
			epoch_err += error;
			if(i % 1000 == 0) printf("i = %d \t idx = %d \t error: %lf\n", i,train_idx, epoch_err/(i+1));
			if(i % 1000 == 0) printf("label = %d , predict = %d \n", train_set[train_idx].label, classify(train_set[train_idx].data, train_idx));
			//if(i % 1000 == 0) printf("error: %lf\n", error);

		}
		
		printf("epoch %d \t error: %lf \t time_on_gpu: %lf \n",epoch, epoch_err/train_cnt, time_taken);
		test_on_train();
		test_on_test();
		printf("-----\n");
		epoch--;
	}

	fprintf(stdout, "\nTime - %lf\n", time_taken);
	
}


// Returns label of given data (0-9)
static unsigned int classify(double data[28][28], int cnt)
{
	double res[10];
	unsigned int max = 0;

	//hipMemcpy(res, l_f.output, sizeof(float) * 10, hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; ++i) {
		res[i] = densed.data1D[i];
	}
	for (int i = 0; i < 10; ++i) {
		if (res[max] < res[i]) {
			max = i;
		}
	}

	return max;
}



// Perform forward propagation of test data
double test_on_test()
{
	int test_acc = 0;

	for (int i = 0; i < test_cnt; ++i) {
		forward_pass(test_set[i].data, i);
		int classify_label = classify(test_set[i].data, i);
		if (classify_label == test_set[i].label) {
			//printf("%d label = %d , predict = %d \n", i, test_set[i].label, classify_label);
			//printimg(test_set[i].data);
			test_acc++;
		}
		

	}

	fprintf(stdout, "test on test accuracy: %.2lf%%\n", double(test_acc) / double(test_cnt) * 100.0);
	return double(test_acc) / double(test_cnt) * 100.0;
}

// Perform forward propagation of test data
double test_on_train()
{
	int test_acc = 0;

	for (int i = 0; i < train_cnt; ++i) {
		forward_pass(train_set[i].data, i);
		int classify_label = classify(train_set[i].data, i);
		if (classify_label == train_set[i].label) {
			//printf("%d label = %d , predict = %d \n", i, test_set[i].label, classify_label);
			//printimg(test_set[i].data);
			test_acc++;
		}
		

	}

	fprintf(stdout, "test on train accuracy: %.2lf%%\n", double(test_acc) / double(train_cnt) * 100.0);
	return double(test_acc) / double(train_cnt) * 100.0;
}
