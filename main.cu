#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE
#include "mnist.h"
#include "layer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <time.h>
#include <cassert>
#include <math.h>
#include <stdio.h>   
#include <stdlib.h> 

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;

int train_onehot[60000][10];
int test_onehot[10000][10];
double input_kernel[26*26*6][2033];
double output_kernel[2033][10];
double input_bias[2033];
double output_bias[10];
double error;
double L_rate = 1.0e-5f;

// Define layers of CNN
static Layer train_input = Layer(1, 28, 28);
static Layer conved = Layer(6, 27, 27);
static Layer maxpooled = Layer(6, 26, 26);
static Layer flattenned = Layer(1, 1, 26*26*6);
static Layer hidden = Layer(1, 1, 2033);
static Layer densed = Layer(1, 1, 10);

static void learn();
static unsigned int classify(double data[28][28], int cnt);
double test_on_train();
double test_on_test();
static double forward_pass(double data[28][28], int cnt);
static double back_pass(int cnt);


static inline void loaddata()
{
	mnist_load("./data/train-images.idx3-ubyte", "./data/train-labels.idx1-ubyte",
		&train_set, &train_cnt);
	mnist_load("./data/t10k-images.idx3-ubyte", "./data/t10k-labels.idx1-ubyte",
		&test_set, &test_cnt);

}

/*
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
*/

static void printimg(double data[28][28]){
	for (int i = 0; i < 28; ++i)
	{
		for (int j = 0; j < 28; ++j)
		{
			printf("%.2lf ", data[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void train_one_hot(mnist_data *data, int output[60000][10]){
	for(int i = 0;i < 60000;i++){
		for(int j = 0;j < 10;j++){
			if(data[i].label == j){
				output[i][j] = 1;
			}
			else{
				output[i][j] = 0;
			}
			
		}
	}
}

void test_one_hot(mnist_data *data, int output[10000][10]){
	for(int i = 0;i < 10000;i++){
		for(int j = 0;j < 10;j++){
			if(data[i].label == j){
				output[i][j] = 1;
			}
			else{
				output[i][j] = 0;
			}
			
		}
	}
}


void generateKernel(){

	for (int i = 0; i < 26 * 26 * 6; i++)
	{
		for (int j = 0; j < 2033; j++)
		{
			input_kernel[i][j] = ( ( rand() + -1 * rand() )%10) * 0.1;

		}

	}

	for (int i = 0; i < 2033; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			output_kernel[i][j] = ( ( rand() + -1 * rand() )%10) * 0.1;

		}

	}

}

void generateBias(){

		for (int i = 0; i < 2033; i++)
		{
			input_bias[i] = ( ( rand() + -1 * rand() )%10) *0.1;

		}

		for (int i = 0; i < 10; i++)
		{
			output_bias[i] = ( ( rand() + -1 * rand() )%10) *0.1;

		}

}



double crossEntropy(int cnt){
	//printf("cross entropy\n");
	double output = 0.0;
	for (int i = 0; i < 10; i++)
	{
		//printf("%.2lf ", densed.data1D[i]);
		if(densed.data1D[i] > 0){
			
			output += log( densed.data1D[i] ) * train_onehot[cnt][i];
		}
		else if(densed.data1D[i] == 0.0){
			output += log( 1.0e-4f ) * train_onehot[cnt][i];
		}
		else{
			printf("%d %d log() error\n", cnt, i);
		}

	}
	//printf("\n");

	output *= -1.0;
	output /= 10.0;

	return output;
	
}

void test_flatten(){
	for (int i = 0; i < maxpooled.C; i++)
	{
		for (int j = 0; j < maxpooled.H; j++)
		{
			for (int k = 0; k < maxpooled.W; k++)
			{
				flattenned.data1D[i * maxpooled.H * maxpooled.W + j * maxpooled.W + k] = maxpooled.data3D[i][j][k];
			}
			
			
		}
		
	}
}

int main(int argc, const  char **argv)
{
	loaddata();
	//printimg(train_set[0].data);
	train_one_hot(train_set, train_onehot);
	test_one_hot(test_set, test_onehot);
	srand (time(NULL));
	generateKernel();
	generateBias();

	printf("----- forward passing takes %lf milliseconds\n", forward_pass(train_set[0].data, 0));
	printf("----- backward passing takes %lf milliseconds\n", back_pass(0));
	// printf("----- forward passing takes %lf milliseconds\n", forward_pass(train_set[3].data, 3));
	// printf("----- backward passing takes %lf milliseconds\n", back_pass(3));
	// printimg(test_set[0].data);
	
	// learn();
	
	return 0;
}

// Forward propagation of a single row in dataset
static double forward_pass(double data[28][28], int cnt){

	clock_t start, end, tmp;
	start = clock();
	
	tmp = clock();
	// printf("origin\n");
	train_input.readInput(train_set[cnt].data);
	// train_input.printData();
	printf("data read in takes %lf milliseconds\n", (double)(clock() - tmp) / ( CLOCKS_PER_SEC/1000));
	
	tmp = clock();
	// printf("convolution\n");
	conved.conv2D(train_input.data2D);
	// conved.printData();
	printf("conv2D layer takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	
	tmp = clock();
	// printf("maxpooling\n");
	maxpooled.maxPooling(conved.data3D);
	// maxpooled.printData();
	printf("maxpooling layer takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	
	tmp = clock();
	// printf("flatten\n");
	test_flatten();
	//flattenned.flatten(maxpooled.data2D);
	// flattenned.printData();
	printf("flatten layer takes %lf millieconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));


	int dummyInt = 1;
	int* dummy;
	if ( hipMalloc((void**)&dummy, sizeof(int) * 1)  != hipSuccess) printf("dummy error\n");
	if ( hipMemcpy(dummy, &dummyInt, sizeof(int) * 1, hipMemcpyHostToDevice) != hipSuccess) printf("dummy cpy error\n");
	hipFree(dummy);
	
	tmp = clock();
	// printf("hidden\n");
	// hidden.in_hidden(flattenned.data1D, input_kernel, input_bias);
	// printf("into hidden layer - calculation takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));

	
	
	double* device_flatten;
	if ( hipMalloc((void**)&device_flatten, sizeof(double) * 676*6)  != hipSuccess) printf("device_flatten error\n");
	if ( hipMemcpy(device_flatten, flattenned.data1D, sizeof(double) * 676*6, hipMemcpyHostToDevice) != hipSuccess) printf("device_flatten cpy error\n");
	double* device_hidden;
	if ( hipMalloc((void**)&device_hidden, sizeof(double) * 2033) != hipSuccess) printf("device_hidden error\n");

	// double* host_input_kernel;
	// hipHostMalloc((void**)&host_input_kernel, sizeof(double) * 676 * 6 * 2033, hipHostMallocDefault);
	// memcpy( &host_input_kernel, &input_kernel,  sizeof(double) * 676 * 6 * 2033);

	double* device_input_kernel;
	if ( hipMalloc((void**)&device_input_kernel, sizeof(double) * 676 * 6 * 2033) != hipSuccess) printf("device_input_kernel error\n");
	if ( hipMemcpy(device_input_kernel, input_kernel, sizeof(double) * 676 * 6 * 2033, hipMemcpyHostToDevice) != hipSuccess) printf("device_input_kernel cpy error\n");
	
	double* device_input_bias;
	if ( hipMalloc((void**)&device_input_bias, sizeof(double) * 2033) != hipSuccess) printf("device_input_bias error\n");
	if ( hipMemcpy(device_input_bias, input_bias, sizeof(double) * 2033, hipMemcpyHostToDevice) != hipSuccess) printf("device_input_bias cpy error\n");
	printf("into hidden layer - memory to GPU takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	
	tmp = clock();
	// GPU_in_hidden<<<2048,2048>>>(device_flatten, device_hidden, host_input_kernel, device_input_bias);
	GPU_in_hidden<<<1024,1024>>>(device_flatten, device_hidden, device_input_kernel, device_input_bias);
	hipDeviceSynchronize();
	printf("into hidden layer - calculation takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	
	tmp = clock();
	if ( hipMemcpy(hidden.data1D, device_hidden, sizeof(double) * 2033, hipMemcpyDeviceToHost) != hipSuccess) printf("device_hidden cpy back error\n");
	printf("into hidden layer - memory to CPU takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	
	hipFree(device_flatten);
	hipFree(device_input_kernel);
	// hipFree(host_input_kernel);
	hipFree(device_input_bias);
	// hidden.printData();
	// printf("into hidden layer takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	

	tmp = clock();
	// printf("densed\n");
	// densed.dense(hidden.data1D, output_kernel, output_bias);
	// printf("dense layer - calculation takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	
	// double* device_hidden;
	// if ( hipMalloc((void**)&device_hidden, sizeof(double) * 2033) != hipSuccess) printf("device_hidden error\n");
	// if ( hipMemcpy(device_hidden, hidden.data1D, sizeof(double) * 2033, hipMemcpyHostToDevice) != hipSuccess) printf("device_output_kernel cpy error\n");
	double* device_densed;
	if ( hipMalloc((void**)&device_densed, sizeof(double) * 10) != hipSuccess) printf("device_densed error\n");
	double* device_output_kernel;
	if ( hipMalloc((void**)&device_output_kernel, sizeof(double) * 10 * 2033) != hipSuccess) printf("device_output_kernel error\n");
	if ( hipMemcpy(device_output_kernel, output_kernel, sizeof(double) * 10 * 2033, hipMemcpyHostToDevice) != hipSuccess) printf("device_output_kernel cpy error\n");
	double* device_output_bias;
	if ( hipMalloc((void**)&device_output_bias, sizeof(double) * 10) != hipSuccess) printf("device_output_bias error\n");
	if ( hipMemcpy(device_output_bias, output_bias, sizeof(double) * 10, hipMemcpyHostToDevice) != hipSuccess) printf("device_output_bias cpy error\n");
	printf("dense layer - memory to GPU takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));

	tmp = clock();
	GPU_dense<<<1,10>>>(device_hidden, device_densed, device_output_kernel, device_output_bias);
	hipDeviceSynchronize();
	printf("dense layer - calculation takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	tmp = clock();
	if ( hipMemcpy(densed.data1D, device_densed, sizeof(double) * 10, hipMemcpyDeviceToHost) != hipSuccess) printf("device_dense cpy back error\n");
	
	hipFree(device_hidden);
	hipFree(device_densed);
	hipFree(device_output_kernel);
	hipFree(device_output_bias);
	// if(cnt % 1000 == 0) densed.printData();
	// densed.printData();
	printf("dense layer - memory to CPU takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	

	tmp = clock();
	error = crossEntropy(cnt);
	// if(cnt % 10000 == 0) printf("%d label = %d , predict = %d \n", cnt, train_set[cnt].label, classify(train_set[cnt].data, cnt));
	//if(cnt < 5) printf("cnt = %d \t error = %lf\n", cnt, error);
	//if(cnt % 1000 == 0) printf("cnt = %d \t error = %lf\n", cnt, error);
	//if(cnt % 10000 == 0) printf("cnt = %d \t error = %lf\n", cnt, error);
	printf("error calculation takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));

	end = clock();

	return ((double) (end - start)) / (CLOCKS_PER_SEC/1000);
}

// Back propagation to update weights
static double back_pass(int cnt)
{
	//printf("back propagation\n");
	clock_t start, end,tmp;

	start = clock();


	tmp = clock();
	//output delta
	double output_delta[10];
	for (int i = 0; i < 10; i++)
	{
		output_delta[i] = densed.data1D[i] - (double)train_onehot[cnt][i];
		
	}
	printf("output delta takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));

	// printf("out delta\n");
	// for (int i = 0; i < 10; i++)
	// {
	// 	printf("%.2lf ", output_delta[i]);

	// }
	// printf("\n");

	//hidden delta
	tmp = clock();
	double hidden_delta[2033];
	double pre_act[2033];
	for (int i = 0; i < 2033; i++)
	{
		pre_act[i] = 0.0;
		for (int j = 0; j < 10; j++)
		{
			pre_act[i] += output_delta[j] * output_kernel[i][j];
		}
		
	}
	printf("pre_act calculation takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	// printf("pre_act\n");
	// for (int i = 0; i < 2033; i++)
	// {
	// 	printf("%.2lf ", pre_act[i]);

	// }
	// printf("\n");



	tmp = clock();
		/*
	for (int i = 0; i < 343; i++)
	{
		partial[i] = 0.0;
		for (int j = 0; j < 676; j++)
		{
			partial[i] += input_kernel[j][i];
		}
			
	}
	*/

	double partial[2033];
	double* device_partial;
	if (hipMalloc((void**)&device_partial, sizeof(double)*2033) != hipSuccess) printf("device_partial error\n");
	double* device_input_kernel;
	if (hipMalloc((void**)&device_input_kernel,sizeof(double) * 676 * 6 * 2033) != hipSuccess) printf("device_input_kernel error\n");
	if ( hipMemcpy(device_input_kernel, input_kernel, sizeof(double)* 676 * 6 * 2033, hipMemcpyHostToDevice) != hipSuccess) printf("device_input_kernel cpy error\n");
	GPU_partial<<<1024,1024>>>(device_input_kernel, device_partial);
	hipDeviceSynchronize();
	if ( hipMemcpy(partial, device_partial, sizeof(double)* 2033, hipMemcpyDeviceToHost) != hipSuccess) printf("device_input_kernel cpy back error\n");
	

	hipFree(device_partial);
	hipFree(device_input_kernel);
	printf("patrial differential takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));


	tmp = clock();
	for (int i = 0; i < 2033; i++)
	{
		
		hidden_delta[i] = pre_act[i] * partial[i];
		
	}
	printf("hidden_delta takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));


	// printf("hidden delta\n");
	// for (int i = 0; i < 2033; i++)
	// {
	// 	printf("%lf ", hidden_delta[i]);
		
	// }
	// printf("\n");
	

	/*
	//input delta
	double input_delta[26*26];
	for (int i = 0; i < 26*26; i++)
	{
		input_delta[i] = 0.0;
		for (int j = 0; j < 343; j++)
		{
			input_delta[i] += flattenned.data1D[i] * hidden_delta[j] * input_kernel[i][j];
		}
		
	}
	*/
	
	//output kernel&bias update
	tmp = clock();
	for (int i = 0; i < 2033; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			output_kernel[i][j] -= L_rate * hidden.data1D[j] * output_delta[j];
		}

	}
	printf("output_kernel update takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	// printf("output kernel\n");
	// for (int i = 0; i < 343; i++)
	// {
	// 	for (int j = 0; j < 10; j++)
	// 	{
	// 		printf("%lf ", output_kernel[i][j]);
	// 	}

	// }
	// printf("\n");

	tmp = clock();
	for (int j = 0; j < 10; j++)
	{
		output_bias[j] -=   L_rate *  output_delta[j];
	}
	printf("output_bias update takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	// printf("output bias\n");
	// for (int i = 0; i < 10; i++)
	// {
	// 	printf("%lf ", output_bias[i]);
	// }
	// printf("\n");

	// printf("input kernel before \n");
	// for (int i = 0; i < 26*26; i++)
	// {
		
	// 	printf("%lf ", input_kernel[i][0]);
	// }
	// printf("\n");

	//input kernel&bias update
	tmp = clock();
	for (int i = 0; i < 26 * 26 * 6; i++)
	{
		for (int j = 0; j < 2033; j++)
		{
			input_kernel[i][j] = input_kernel[i][j] - L_rate * flattenned.data1D[i] * hidden_delta[j];
		}

	}
	printf("input_kernel update takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));
	// printf("input kernel\n");
	// for (int i = 0; i < 26*26 * 6; i++)
	// {
		
	// 	printf("%lf ", input_kernel[i][123]);
	// }
	// printf("\n");

	tmp = clock();
	for (int j = 0; j < 2033; j++)
	{
		input_bias[j] -=   L_rate *  hidden_delta[j];
	}
	printf("input_bias update takes %lf milliseconds\n", (double) (clock() - tmp) / (CLOCKS_PER_SEC/1000));

	end = clock();
	return ((double) (end - start)) / (CLOCKS_PER_SEC/1000);
}

static void learn()
{
	
	int epoch = 3;
	double time_taken = 0.0;
	int patience = 2;
	double patience_factor = 0.2;
	int patience_idx = 0;
	bool *patience_test = new bool(patience);
	double last_epoch_err;
	train_cnt = 10000;
	
	fprintf(stdout ,"Learning \n");

	for (int epoch_cnt = 0; epoch_cnt < epoch; epoch_cnt++)
	{
		double epoch_err = 0.0;
		int train_idx;
		printf("epoch %d\n", epoch_cnt);
		for (int i = 0; i < train_cnt; i++) {
			
			//printf("forward passing\n");
			train_idx = rand()%train_cnt;
			time_taken += forward_pass(train_set[train_idx].data, train_idx);
			time_taken += back_pass(train_idx);
			epoch_err += error;
			if(i % 1000 == 0) printf("i = %d \t idx = %d \t error: %lf\n", i,train_idx, epoch_err/(i+1));
			if(i % 1000 == 0) printf("label = %d , predict = %d \n", train_set[train_idx].label, classify(train_set[train_idx].data, train_idx));
			//if(i % 1000 == 0) printf("error: %lf\n", error);

		}

		printf("epoch %d \t error: %lf \t time_on_gpu: %lf \n",epoch_cnt, epoch_err/train_cnt, time_taken);

		if(epoch_cnt == 0) last_epoch_err = epoch_err/train_cnt;
		patience_test[ (patience_idx+1) % patience ] = (last_epoch_err - epoch_err/train_cnt) >= 0? false:true;
		last_epoch_err = epoch_err/train_cnt;
		if (epoch_cnt > 1 && patience_test[0] && patience_test[1]) L_rate *= patience_factor;
	
		test_on_train();
		//test_on_test();
		printf("-----\n");

	}

	fprintf(stdout, "\nTime - %lf\n", time_taken);
	
}


// Returns label of given data (0-9)
static unsigned int classify(double data[28][28], int cnt)
{
	double res[10];
	unsigned int max = 0;

	//hipMemcpy(res, l_f.output, sizeof(float) * 10, hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; ++i) {
		res[i] = densed.data1D[i];
	}
	for (int i = 0; i < 10; ++i) {
		if (res[max] < res[i]) {
			max = i;
		}
	}

	return max;
}



// Perform forward propagation of test data
double test_on_test()
{
	int test_acc = 0;

	for (int i = 0; i < test_cnt; ++i) {
		forward_pass(test_set[i].data, i);
		int classify_label = classify(test_set[i].data, i);
		if (classify_label == test_set[i].label) {
			//printf("%d label = %d , predict = %d \n", i, test_set[i].label, classify_label);
			//printimg(test_set[i].data);
			test_acc++;
		}
		

	}

	fprintf(stdout, "test on test accuracy: %.2lf%%\n", double(test_acc) / double(test_cnt) * 100.0);
	return double(test_acc) / double(test_cnt) * 100.0;
}

// Perform forward propagation of test data
double test_on_train()
{
	int test_acc = 0;

	for (int i = 0; i < train_cnt; ++i) {
		forward_pass(train_set[i].data, i);
		int classify_label = classify(train_set[i].data, i);
		if (classify_label == train_set[i].label) {
			//printf("%d label = %d , predict = %d \n", i, test_set[i].label, classify_label);
			//printimg(test_set[i].data);
			test_acc++;
		}
		

	}

	fprintf(stdout, "test on train accuracy: %.2lf%%\n", double(test_acc) / double(train_cnt) * 100.0);
	return double(test_acc) / double(train_cnt) * 100.0;
}
